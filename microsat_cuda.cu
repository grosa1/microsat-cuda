#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <sys/stat.h>
#include <dirent.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <string>

 #define DB_MAX_MEM 412000;
//#define DB_MAX_MEM 100000;
#define CLAUSE_LEARN_MAX_MEM 100000;
// #define INITIAL_MAX_LEMMAS 100; //initial max learnt clauses
#define INITIAL_MAX_LEMMAS 2000; //initial max learnt clauses

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

struct solver { // The variables in the struct are described in the allocate procedure
	int* DB, nVars, nClauses, mem_used, mem_fixed, mem_max, maxLemmas, nLemmas, * buffer, nConflicts, * model,
		* reason, * falseStack, * _false, * first, * forced, * processed, * assigned, * next, * prev, head, res, fast, slow,
		result, file_id;
};

typedef struct {
	int files_count;
	double parse_time;
	double init_time;
	double solve_time;
	double tot_time;
} Metrics;

enum { END = -9, UNSAT = 0, SAT = 1, MARK = 2, IMPLIED = 6 };

__device__
int* getMemory(struct solver* S, int mem_size) {                  // Allocate memory of size mem_size
	if (S->mem_used + mem_size > S->mem_max) {                       // In case the code is used within a code base
		printf("c out of memory\n");
		return 0;
	}
	int* store = (S->DB + S->mem_used);                              // Compute a pointer to the new memory location
	S->mem_used += mem_size;                                         // Update the size of the used memory
	return store;
}                                                  // Return the pointer

__device__
void unassign(struct solver* S, int lit) { S->_false[lit] = 0; }   // Unassign the literal

__device__
void restart(struct solver* S) {                                  // Perform a restart (i.e., unassign all variables)
	while (S->assigned > S->forced) unassign(S, *(--S->assigned));  // Remove all unforced false lits from falseStack
	S->processed = S->forced;
}                                      // Reset the processed pointer

__device__
void assign(struct solver* S, int* reason, int forced) {          // Make the first literal of the reason true
	int lit = reason[0];                                             // Let lit be the first ltieral in the reason
	S->_false[-lit] = forced ? IMPLIED : 1;                           // Mark lit as true and IMPLIED if forced
	*(S->assigned++) = -lit;                                         // Push it on the assignment stack
	S->reason[abs(lit)] = 1 + (int)((reason)-S->DB);               // Set the reason clause of lit
	S->model[abs(lit)] = (lit > 0);
}                              // Mark the literal as true in the model

__device__
void addWatch(struct solver* S, int lit, int mem) {               // Add a watch pointer to a cfor entry function '_Z5solveP6solver' containing lit
	S->DB[mem] = S->first[lit]; S->first[lit] = mem;
}               // By updating the database afor entry function '_Z5solveP6solver'e pointers

__device__
int* addClause(struct solver* S, int* buffer, int size, int irr) {    // Adds a clause stored in *in of size size
	int i, used = S->mem_used;                                       // Store a pointer to the beginning of the clause
	int* clause = getMemory(S, size + 3) + 2;                       // Allocate memory for the clause in the database
	if (size > 1) {
		addWatch(S, buffer[0], used);                    // If the clause is not unit, then add
		addWatch(S, buffer[1], used + 1);
	}                  // Two watch pointers to the datastructure
	for (i = 0; i < size; i++) clause[i] = buffer[i]; clause[i] = 0;     // Copy the clause from the buffer to the database
	if (irr) S->mem_fixed = S->mem_used; else S->nLemmas++;          // Update the statistics
	return clause;
}                                                 // Return the pointer to the clause is the database

__device__
void reduceDB(struct solver* S, int k) {                     // Removes "less useful" lemmas from DB
	//printf("Start reduceDB function\n");
	while (S->nLemmas > S->maxLemmas) S->maxLemmas += 300;      // Allow more lemmas in the future
	S->nLemmas = 0;                                             // Reset the number of lemmas

	int i; for (i = -S->nVars; i <= S->nVars; i++) {            // Loop over the variables
		if (i == 0) continue; int* watch = &S->first[i];          // Get the pointer to the first watched clause
		while (*watch != END)                                     // As long as there are watched clauses
			if (*watch < S->mem_fixed) watch = (S->DB + *watch);    // Remove the watch if it points to a lemma
			else                      *watch = S->DB[*watch];
	}  // Otherwise (meaning an input clause) go to next watch

	int old_used = S->mem_used; S->mem_used = S->mem_fixed;     // Virtually remove all lemmas
	for (i = S->mem_fixed + 2; i < old_used; i += 3) {          // While the old memory contains lemmas
		int count = 0, head = i;                                  // Get the lemma to which the head is pointing
		while (S->DB[i]) {
			int lit = S->DB[i++];                  // Count the number of literals
			if ((lit > 0) == S->model[abs(lit)]) count++;
		}        // That are satisfied by the current model
		if (count < k) addClause(S, S->DB + head, i - head, 0);
	}
}  // If the latter is smaller than k, add it back

__device__
void bump(struct solver* S, int lit) {                       // Move the variable to the front of the decision list
	//printf("Start bump function\n");
	if (S->_false[lit] != IMPLIED) {
		S->_false[lit] = MARK;       // MARK the literal as involved if not a top-level unit
		int var = abs(lit); if (var != S->head) {                // In case var is not already the head of the list
			S->prev[S->next[var]] = S->prev[var];                   // Update the prev link, and
			S->next[S->prev[var]] = S->next[var];                   // Update the next link, and
			S->next[S->head] = var;                                 // Add a next link to the head, and
			S->prev[var] = S->head; S->head = var;
		}
	}
}            // Make var the new head

__device__
int implied(struct solver* S, int lit) {                  // Check if lit(eral) is implied by MARK literals
//	printf("Start implied function\n");
	if (S->_false[lit] > MARK) return (S->_false[lit] & MARK); // If checked before return old result
	if (!S->reason[abs(lit)]) return 0;                     // In case lit is a decision, it is not implied
	int* p = (S->DB + S->reason[abs(lit)] - 1);             // Get the reason of lit(eral)
	while (*(++p))                                           // While there are literals in the reason
		if ((S->_false[*p] ^ MARK) && !implied(S, *p)) {       // Recursively check if non-MARK literals are implied
			S->_false[lit] = IMPLIED - 1; return 0;
		}             // Mark and return not implied (denoted by IMPLIED - 1)
	S->_false[lit] = IMPLIED; return 1;
}                     // Mark and return that the literal is implied

__device__
int* analyze(struct solver* S, int* clause) {         // Compute a resolvent from falsified clause
//	printf("Start analyze\n");
	S->res++; S->nConflicts++;                           // Bump restarts and update the statistic
	while (*clause) bump(S, *(clause++));               // MARK all literals in the falsified clause
	while (S->reason[abs(*(--S->assigned))]) {          // Loop on variables on falseStack until the last decision
		if (S->_false[*S->assigned] == MARK) {              // If the tail of the stack is MARK
			int* check = S->assigned;                        // Pointer to check if first-UIP is reached
			while (S->_false[*(--check)] != MARK)             // Check for a MARK literal before decision
				if (!S->reason[abs(*check)]) goto build;       // Otherwise it is the first-UIP so break
			clause = S->DB + S->reason[abs(*S->assigned)];  // Get the reason and ignore first literal
			while (*clause) bump(S, *(clause++));
		}         // MARK all literals in reason
		unassign(S, *S->assigned);
	}                      // Unassign the tail of the stack

build:; int size = 0, lbd = 0, flag = 0;             // Build conflict clause; Empty the clause buffer
	int* p = S->processed = S->assigned;                 // Loop from tail to front
	while (p >= S->forced) {                             // Only literals on the stack can be MARKed
		if ((S->_false[*p] == MARK) && !implied(S, *p)) {  // If MARKed and not implied
			S->buffer[size++] = *p; flag = 1;
		}              // Add literal to conflict clause buffer
		if (!S->reason[abs(*p)]) {
			lbd += flag; flag = 0; // Increase LBD for a decision with a true flag
			if (size == 1) S->processed = p;
		}               // And update the processed pointer
		S->_false[*(p--)] = 1;
	}                            // Reset the MARK flag for all variables on the stack

	S->fast -= S->fast >> 5; S->fast += lbd << 15;      // Update the fast moving average
	S->slow -= S->slow >> 15; S->slow += lbd << 5;      // Update the slow moving average

	while (S->assigned > S->processed)                   // Loop over all unprocessed literals
		unassign(S, *(S->assigned--));                    // Unassign all lits between tail & head
	unassign(S, *S->assigned);                          // Assigned now equal to processed
	S->buffer[size] = 0;                                 // Terminate the buffer (and potentially print clause)
	return addClause(S, S->buffer, size, 0);
}          // Add new conflict clause to redundant DB

__device__
int propagate(struct solver* S) {                  // Performs unit propagation
	int forced = S->reason[abs(*S->processed)];      // Initialize forced flag
	while (S->processed < S->assigned) {              // While unprocessed false literals
		int lit = *(S->processed++);                    // Get first unprocessed literal
		int* watch = &S->first[lit];                    // Obtain the first watch pointer
		while (*watch != END) {                         // While there are watched clauses (watched by lit)
			int i, unit = 1;                              // Let's assume that the clause is unit
			int* clause = (S->DB + *watch + 1);	    // Get the clause from DB
			if (clause[-2] == 0) clause++;              // Set the pointer to the first literal in the clause
			if (clause[0] == lit) clause[0] = clause[1]; // Ensure that the other watched literal is in front
			for (i = 2; unit && clause[i]; i++)           // Scan the non-watched literals
				if (!S->_false[clause[i]]) {                 // When clause[i] is not false, it is either true or unset
					clause[1] = clause[i]; clause[i] = lit;   // Swap literals
					int store = *watch; unit = 0;             // Store the old watch
					*watch = S->DB[*watch];                   // Remove the watch from the list of lit
					//printf("add watch\n");
					addWatch(S, clause[1], store);
				}         // Add the watch to the list of clause[1]
			if (unit) {                                   // If the clause is indeed unit

				//printf("unit\n");
				clause[1] = lit; watch = (S->DB + *watch);  // Place lit at clause[1] and update next watch
				if (S->_false[-clause[0]]) continue;        // If the other watched literal is satisfied continue
				if (!S->_false[clause[0]]) {                // If the other watched literal is falsified,
					assign(S, clause, forced);
				}             // A unit clause is found, and the reason is set
				else {
					if (forced) {		// Found a root level conflict -> UNSAT
						//S->result = 0;
						return UNSAT;
					}
					int* lemma = analyze(S, clause);	    // Analyze the conflict return a conflict clause
					if (!lemma[1]) forced = 1;                // In case a unit clause is found, set forced flag
					assign(S, lemma, forced); break;
				}
			}
		}
	} // Assign the conflict clause as a unit

	if (forced) S->forced = S->processed;	            // Set S->forced if applicable
	//S->result = 1;
	return SAT;
}	                                    // Finally, no conflict was found

__global__
void solve(struct solver** multi_s) {    // Determine satisfiability
	struct solver* S = multi_s[threadIdx.x];

	int decision = S->head; S->res = 0;                               // Initialize the solver
	for (;;) {                                                        // Main solve loop
		int old_nLemmas = S->nLemmas;                                   // Store nLemmas to see whether propagate adds lemmas
		int res = propagate(S);
		if (res == UNSAT) {
			printf("file_%d=UNSAT,vars=%i,clauses=%i,mem=%i,conflicts=%i,lemmas=%i\n", S->file_id,S->nVars,S->nClauses,S->mem_used,S->nConflicts,S->maxLemmas);
			multi_s[threadIdx.x]->result = UNSAT;
			//printf("result -->", S->result);
			return;
		}                                                               // Propagation returns UNSAT for a root level conflict

		if (S->nLemmas > old_nLemmas) {                                 // If the last decision caused a conflict
			decision = S->head;                                           // Reset the decision heuristic to head
			if (S->fast > (S->slow / 100) * 125) {                        // If fast average is substantially larger than slow average
	  //        printf("c restarting after %i conflicts (%i %i) %i\n", S->res, S->fast, S->slow, S->nLemmas > S->maxLemmas);
				S->res = 0; S->fast = (S->slow / 100) * 125; restart(S);   // Restart and update the averages
				if (S->nLemmas > S->maxLemmas) reduceDB(S, 6);
			}
		}         // Reduce the DB when it contains too many lemmas

		while (S->_false[decision] || S->_false[-decision]) {             // As long as the temporay decision is assigned
			decision = S->prev[decision];
		}
		//printf("decision: %d \n", decision);                               // Replace it with the next variable in the decision list
		if (decision == 0) {
			printf("file_%d=SAT,vars=%i,clauses=%i,mem=%i,conflicts=%i,lemmas=%i\n", S->file_id,S->nVars,S->nClauses,S->mem_used,S->nConflicts,S->maxLemmas);
			multi_s[threadIdx.x]->result = SAT;
			//printf("result -->", S->result );
			return;                                  // If the end of the list is reached, then a solution is found
		}
		decision = S->model[decision] ? decision : -decision;           // Otherwise, assign the decision variable based on the model
		S->_false[-decision] = 1;                                        // Assign the decision literal to true (change to IMPLIED-1?)
		*(S->assigned++) = -decision;                                   // And push it on the assigned stack
		decision = abs(decision); S->reason[decision] = 0;
	}
}          // Decisions have no reason clauses

__global__
void init(struct solver* S, int* dev_elements, int nElements, int nVars, int nClauses, int* db, int*file_id) {                            // Parse the formula and initialize
	int verb = 0;
	if (verb)("\n init \n");
	S->file_id = *file_id;
	S->nVars=nVars;
	if (verb)printf("\n S->nVars -> %d\n", S->nVars);
	S->nClauses= nClauses;
	if (verb)printf("\n S->nClauses -> %d\n", S->nClauses);

	//S->mem_max = 100000;            // Set the initial maximum memory
	S->mem_max = DB_MAX_MEM;            // Set the initial maximum memory
	if (verb)printf("\n S->mem_max -> %d\n", S->mem_max);
	S->mem_used = 0;                  // The number of integers allocated in the DB
	if (verb)printf("\n S->mem_used -> %d\n", S->mem_used);
	S->nLemmas = 0;                  // The number of learned clauses -- redundant means learned
	if (verb)printf("\n S->nLemmas -> %d\n", S->nLemmas);
	S->nConflicts = 0;                  // Under of conflicts which is used to updates scores
	if (verb)printf("\n S->nConflicts -> %d\n", S->nConflicts);
	S->maxLemmas = INITIAL_MAX_LEMMAS;               // Initial maximum number of learnt clauses
	if (verb)printf("\n S->maxLemmas -> %d\n", S->maxLemmas);
	//S->fast = S->slow = 1 << 24;            // Initialize the fast and slow moving averages
	S->fast = S->slow = CLAUSE_LEARN_MAX_MEM;            // Initialize the fast and slow moving averages
	if (verb)printf("\n S->fast -> %d\n", S->fast);
	if (verb)printf("\n S->slow -> %d\n", S->slow);
	S->result = -1;
	if (verb)printf("\n S->result -> %d\n", S->result);

	S->DB = db;
	if (verb)printf("\n S->DB -> %d \n", S->DB);

	S->model = getMemory(S, S->nVars + 1); // Full assignment of the (Boolean) variables (initially set to false)
	if (verb)printf("\n S->model -> %d \n", S->model);

	S->next = getMemory(S, S->nVars + 1); // Next variable in the heuristic order
	if (verb)printf("\n S->next -> %d \n", S->next);

	S->prev = getMemory(S, S->nVars + 1); // Previous variable in the heuristic order
	if (verb)printf("\n S->prev -> %d \n", S->prev);

	S->buffer = getMemory(S, S->nVars); // A buffer to store a temporary clause
	if (verb)printf("\n S->buffer -> %d \n", S->buffer);

	S->reason = getMemory(S, S->nVars + 1); // Array of clauses
	if (verb)printf("\n S->reason -> %d \n", S->reason);

	S->falseStack = getMemory(S, S->nVars + 1); // Stack of falsified literals -- this pointer is never changed
	if (verb)printf("\n S->falseStack -> %d \n", S->falseStack);

	S->forced = S->falseStack;      // Points inside *falseStack at first decision (unforced literal)
	if (verb)printf("\n S->forced -> %d \n", S->forced);
	S->processed = S->falseStack;      // Points inside *falseStack at first unprocessed literal
	if (verb)printf("\n S->processed -> %d \n", S->processed);
	S->assigned = S->falseStack;      // Points inside *falseStack at last unprocessed literal
	if (verb)printf("\n S->assigned -> %d \n", S->assigned);

	S->_false = getMemory(S, 2 * S->nVars + 1);
	S->_false += S->nVars; // Labels for variables, non-zero means false
	if (verb)printf("\n S->_false -> %d \n", S->_false);

	S->first = getMemory(S, 2 * S->nVars + 1);
	S->first += S->nVars; // Offset of the first watched clause
	if (verb)printf("\n S->first -> %d \n", S->first);

	S->DB[S->mem_used++] = 0;            // Make sure there is a 0 before the clauses are loaded.
	if (verb)printf("\n S->DB[S->mem_used] -> %d \n", S->DB[S->mem_used-1]);

	if (verb)printf("\n elements \n");
	int i; for (i = 1; i <= S->nVars; i++) {							// Initialize the main datastructes:
		S->prev[i] = i - 1;
		if (verb)printf("\n S->prev[i] -> %d \n", S->prev[i]);

		S->next[i - 1] = i;
		if (verb)printf("\n S->next[i-1] -> %d \n", S->next[i - 1]);

		S->model[i] = S->_false[-i] = S->_false[i] = 0;
		if (verb)printf("\n S->model[i] -> %d \n", S->model[i]);
		if (verb)printf("\n S->_false[i] -> %d \n", S->_false[i]);
		if (verb)printf("\n S->_false[-i] -> %d \n", S->_false[-i]);

		S->first[i] = S->first[-i] = END;						// and first (watch pointers).
		if (verb)printf("\n S->first[i] -> %d \n", S->first[i]);
		if (verb)printf("\n S->first[i] -> %d \n", S->first[-i]);
		S->head = S->nVars;												// Initialize the head of the double-linked list
		if (verb)printf("\n S->head -> %d \n", S->head);
	}


	int nZeros = S->nClauses, size = 0;                      // Initialize the number of clauses to read
	if (verb)printf("\n nZeros -> %d \n", nZeros);
	for (int i = 0; i < nElements;i++) {                                     // While there are elements
		int lit = 0;
		lit= dev_elements[i];
		if (verb)printf("\n lit -> %d \n", lit);

		if (!lit) {                                            // If reaching the end of the clause
			if (verb)printf("\n addClause \n");
			int* clause = addClause(S, S->buffer, size, 1);     // Then add the clause to data_base
			if (verb)printf("\n clause -> %d \n", clause);

			if (verb)printf("\n size -> %d \n", size);
			if (verb)printf("\n S->_false[clause[0]] -> %d \n", S->_false[clause[0]]);
			if (!size || ((size == 1) && S->_false[clause[0]])) {  // Check for empty clause or conflicting unit

				printf("\n + UNSAT + \n");
				S->result = 1;
				return;
			}                                     // If either is found return UNSAT
			if ((size == 1) && !S->_false[-clause[0]]) {          // Check for a new unit
				if (verb)printf("\n assign \n");
				assign(S, clause, 1);
			}                           // Directly assign new units (forced = 1)
			size = 0; --nZeros;
		}
		else S->buffer[size++] = lit;
	}
	//printf("\n INITIALIZED \n");
}                                            // Return that no conflict was observed

__host__
static void read_until_new_line(FILE* input) {
	int ch;
	while ((ch = getc(input)) != '\n')
		if (ch == EOF) { printf("parse error: unexpected EOF"); exit(1); }
}

 int main(int argc, char** argv) {
	//char* directory = "C://microsat//sat";
	char* directory = argv[1];
	int num_file =0;
	int nVars = 0;
	int nClauses = 0;
	Metrics exec_metrics = {0, 0, 0, 0, 0};

	int db_max_mem =DB_MAX_MEM;
	int clause_learn_max_mem = CLAUSE_LEARN_MAX_MEM;
	int initial_max_mem =  INITIAL_MAX_LEMMAS;
    printf("DB_MAX_MEM: %d\n", db_max_mem);
    printf("CLAUSE_LEARN_MAX_MEM: %d\n", clause_learn_max_mem);
    printf("INITIAL_MAX_LEMMAS: %d\n", initial_max_mem);
    
	clock_t start, end;
	printf(" Start\n");
	start = clock();
    
	DIR* dirp;
	struct dirent* entry;
	dirp = opendir(directory);
	while ((entry = readdir(dirp)) != NULL) {
		if (entry->d_type == DT_REG) { /* If the entry is a regular file */
			num_file++;
		}
	}
	closedir(dirp);
	exec_metrics.files_count = num_file;
	//printf(" num file -> %d\n",num_file);

	solver** h_multi_struct;
	h_multi_struct = (solver**)malloc(num_file * sizeof(solver*));
	solver** d_multi_struct;
	gpuErrchk(hipMalloc((void**)&d_multi_struct, num_file * sizeof(solver*)));


	if (NULL == (dirp = opendir(directory)))
	{
		printf("Error : Failed to open input directory \n");
		return 1;
	}

	clock_t start_parse = clock();

	int count = 0;
	while ((entry = readdir(dirp)))
	{
		if (!strcmp(entry->d_name, "."))
			continue;
		if (!strcmp(entry->d_name, ".."))
			continue;

		char path[100] = ""; //TODO: magic number
		strcpy(path, directory);
		strcat(path, "//");
		strcat(path, entry->d_name);
		printf("file_%d=%s\n", count, entry->d_name);

		FILE* input = fopen(path, "r");
		if (input == NULL)
		{
			printf("Error : Failed to open entry file \n");
			fclose(input);
			return 1;
		}

		struct solver* dev_s;
		gpuErrchk(hipMalloc((void**)&dev_s, sizeof(solver)));

		int* db;
		//int mem = 100000; //TODO: allocazione dinamica della memoria
		int mem = DB_MAX_MEM; //TODO: allocazione dinamica della memoria
		gpuErrchk(hipMalloc((void**)&db, sizeof(int) * mem));

		struct stat st;
		stat(path, &st);
		int size = st.st_size;
		//printf("\n size -> %d\n", size);

		int* buffer = 0;
		buffer = (int*)malloc(size * sizeof(int));

		/********* FILE PARSER **************/
		int tmp;
		while ((tmp = getc(input)) == 'c') read_until_new_line(input);
		ungetc(tmp, input);
		do {
			tmp = fscanf(input, " p cnf %i %i \n", &nVars, &nClauses);
			if (tmp > 0 && tmp != EOF) break; tmp = fscanf(input, "%*s\n");
		} while (tmp != 2 && tmp != EOF);

		int nElements = 0;
		do {
			int ch = getc(input);
			if(ch == '\%') break; //we have % as EOF in some dimacs files
			if ( ch == ' ' || ch == '\n') continue;
			if (ch == 'c') { read_until_new_line(input); continue; }
			ungetc(ch, input);
			int lit = 0;
			tmp = fscanf(input, " %i ", &lit);
			buffer[nElements] = lit;
			//printf("%d ", lit);
			nElements++;
		} while (tmp != EOF);

		nElements--; // TO CHECK
		int* elements = 0;
		elements = (int*)malloc(nElements * sizeof(int));
		for (int i = 0; i < nElements; i++) {
			elements[i] = buffer[i];
		}
		fclose(input);
		/********* FILE PARSER **************/

		int* dev_file_id;
		hipMalloc((void**)&dev_file_id, sizeof(int));
		hipMemcpy(dev_file_id, &count, sizeof(int), hipMemcpyHostToDevice);

		int* dev_elements;
		hipMalloc((void**)&dev_elements, nElements * sizeof(int));
		hipMemcpy(dev_elements, elements, nElements * sizeof(int), hipMemcpyHostToDevice);

		free(buffer);
		free(elements);

		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024);

		//printf("\n INIT \n");
		hipEvent_t d_start_init, d_stop_init;
		hipEventCreate(&d_start_init);
		hipEventCreate(&d_stop_init);

		hipEventRecord(d_start_init, 0);
		init << <1, 1 >> > (dev_s,dev_elements,nElements,nVars,nClauses,db,dev_file_id);
		hipEventRecord(d_stop_init, 0);
		hipEventSynchronize(d_stop_init);

		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, d_start_init, d_stop_init); // that's our time!
		exec_metrics.init_time += elapsedTime;
		// Clean up:
		hipEventDestroy(d_start_init);
		hipEventDestroy(d_stop_init);

		//printf("parsing_file -> %s\n", entry->d_name);
		//printf("device_time -> %f s\n", elapsedTime / 1000000);
		//exec_metrics.init_time += elapsedTime / 1000000;

		hipDeviceSynchronize();

		//temp
		//printf("\n dev_s -> %p\n",dev_s);
		h_multi_struct[count] = dev_s;
		count++;
	}
/*********** end init and parse ***********/
exec_metrics.parse_time = (clock() - start_parse);


	hipMemcpy(d_multi_struct, h_multi_struct, num_file * sizeof(solver*), hipMemcpyHostToDevice);
	//temp end

	printf("\n SOLVE \n");
	hipEvent_t d_start, d_stop;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	hipEventRecord(d_start, 0);
	solve<< <1, num_file >> > (d_multi_struct);
	hipEventRecord(d_stop, 0);
	hipEventSynchronize(d_stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d_start, d_stop); // that's our time!
	// Clean up:
	hipEventDestroy(d_start);
	hipEventDestroy(d_stop);

	//printf("\n total solve time -> %f s\n", elapsedTime / 1000000);
	exec_metrics.solve_time = elapsedTime;
	hipDeviceSynchronize();

	hipDeviceReset();

	end = clock();
	//printf("\n total time: %f s\n", (float)(end - start) / 1000000);
	exec_metrics.tot_time = (float)(end - start);
	printf("\n+++ metrics (ms)+++\nfiles count: %d\nparse time: %f\ncuda init time: %f\ncuda solve time: %f\ntot time: %f\n\n", exec_metrics.files_count, exec_metrics.parse_time/CLOCKS_PER_SEC, exec_metrics.init_time/1000, exec_metrics.solve_time/1000, exec_metrics.tot_time/CLOCKS_PER_SEC);
	//printf ("c statistics of %s: mem: %i conflicts: %i max_lemmas: %i\n", argv[1], S.mem_used, S.nConflicts, S.maxLemmas);
	//printf("\n END \n");
	return 0;
}
